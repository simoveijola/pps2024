#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#if USE_MPI
#include <mpi.h>
#endif

#include "errchk.h"
#include "reduce.cuh"

#define NUM_SAMPLES (1024) // (1024)


const size_t random_seed = 123456789;



#define MAX_COUNT ((1 * 1024 * 1024) / sizeof(int))
#define ARRAY_SIZE(arr) (sizeof(arr) / sizeof(arr[0]))

static int
modelreduce(const int* arr, const size_t count)
{
  ERRCHK(count > 0);
  int max = arr[0];

  for (int i = 1; i < count; ++i)
    max = arr[i] > max ? arr[i] : max;

  return max;
}

static void
write_to_file(const char* path, const char* str)
{
  FILE* fp = fopen(path, "w");
  ERRCHK(fp);

  fprintf(fp, "%s\n", str);

  fclose(fp);
}
int*
get_arr(const size_t count)
{
	return (int*)calloc(count, sizeof(int));
}

int
test(int* arr, const size_t count, const int pid)
{
    const size_t tests[] = {0, rand() % count, count - 1};

    for (size_t j = 0; j < ARRAY_SIZE(tests); ++j) {
      arr[tests[j]]       = j + 1;
      const int model     = modelreduce(arr, count);
      const int candidate = reduce(arr, count);

      if (!pid) {
        printf("Position: %*lu, Model: %d, Candidate: %d, Correct? %s\n", 6,
               tests[j], model, candidate, model == candidate ? "Yes" : "No");
        fflush(stdout);

        if (model != candidate) {
          fprintf(stderr,
                  "Failure at: Position: %*lu, Model: %d, Candidate: %d, "
                  "Correct? %s\n",
                  6, tests[j], model, candidate,
                  model == candidate ? "Yes" : "No");
          return 1;
        }
      }
    }
    return 0;
}


int
main(int argc, char* argv[])
{
  char* outfile = NULL;
  if (argc > 1)
    outfile = argv[1];

#if USE_MPI
  MPI_Init(NULL, NULL);
  int nprocs, pid;
  MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
  MPI_Comm_rank(MPI_COMM_WORLD, &pid);
#else
  int num_devices;
  ERRCHK_CUDA(hipGetDeviceCount(&num_devices));
  const int nprocs = num_devices;
  const int pid    = 0;
#endif

  srand(random_seed);
  for (size_t i = 0; i < NUM_SAMPLES; ++i) {
    // Set count a multiple of nprocs/devices for simplicity
    const size_t count = nprocs * (1 + (rand() % (MAX_COUNT / nprocs)));
    int* arr           = get_arr(count);
    int failed = test(arr,count,pid); 
    free(arr);
#if USE_MPI
    MPI_Bcast(&failed, 1, MPI_INT, 0, MPI_COMM_WORLD);
#endif
    if(failed) {
          if (outfile && pid == 0)
            write_to_file(outfile, "FAILURE");
#if USE_MPI
          MPI_Barrier(MPI_COMM_WORLD);
          MPI_Finalize();
#endif
        printf("pid %d exit failure\n", pid);
	fflush(stdout);
	return EXIT_FAILURE;
    }
  }

#if USE_MPI
  MPI_Barrier(MPI_COMM_WORLD);
  MPI_Finalize();
#endif

  if (outfile && pid == 0)
    write_to_file(outfile, "OK");

  return EXIT_SUCCESS;
}
